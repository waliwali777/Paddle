
#include <hip/hip_runtime.h>
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_XPU_KP
#include "paddle/phi/kernels/bitwise_kernel.h"

// #include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/xpu/xpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/bitwise_functors.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
namespace phi {

#define DEFINE_BITWISE_KERNEL(op_type)                      \
  template <typename T, typename Context>                   \
  void Bitwise##op_type##Kernel(const Context& dev_ctx,     \
                                const DenseTensor& x,       \
                                const DenseTensor& y,       \
                                DenseTensor* out) {         \
    dev_ctx.template Alloc<T>(out);                         \
    funcs::Bitwise##op_type##Functor<T> func;               \
    std::vector<const DenseTensor*> ins = {&x, &y};         \
    std::vector<DenseTensor*> outs = {out};                 \
    funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>( \
        dev_ctx, ins, &outs, -1, func);                     \
  }

DEFINE_BITWISE_KERNEL(And)
DEFINE_BITWISE_KERNEL(Or)
DEFINE_BITWISE_KERNEL(Xor)
#undef DEFINE_BITWISE_KERNEL

template <typename T, typename Context>
void BitwiseNotKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  funcs::BitwiseNotFunctor<T> func;
  funcs::BroadcastKernel<ElementwiseType::kUnary, T, T>(
      dev_ctx, ins, &outs, -1, func);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    bitwise_and, KPS, ALL_LAYOUT, phi::BitwiseAndKernel, int, bool) {}
PD_REGISTER_KERNEL(
    bitwise_or, KPS, ALL_LAYOUT, phi::BitwiseOrKernel, int, bool) {}
PD_REGISTER_KERNEL(
    bitwise_xor, KPS, ALL_LAYOUT, phi::BitwiseXorKernel, int, bool) {}
PD_REGISTER_KERNEL(
    bitwise_not, KPS, ALL_LAYOUT, phi::BitwiseNotKernel, int, bool) {}
#endif
