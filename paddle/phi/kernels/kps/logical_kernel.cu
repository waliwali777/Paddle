
#include <hip/hip_runtime.h>
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitation

#ifdef PADDLE_WITH_XPU_KP
#include "paddle/phi/kernels/logical_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/logical_functor.h"

namespace phi {

#define DEFINE_LOGICAL_BINARY_KERNEL(type)                               \
  template <typename T, typename Context>                                \
  void Logical##type##Kernel(const Context& dev_ctx,                     \
                             const DenseTensor& x,                       \
                             const DenseTensor& y,                       \
                             DenseTensor* out) {                         \
    using InT = typename funcs::Logical##type##Functor<T>::ELEMENT_TYPE; \
    using OutT = bool;                                                   \
    dev_ctx.template Alloc<bool>(out);                                   \
    funcs::Logical##type##Functor<T> binary_func;                        \
    std::vector<const DenseTensor*> ins = {&x, &y};                      \
    std::vector<DenseTensor*> outs = {out};                              \
    funcs::BroadcastKernel<ElementwiseType::kBinary, InT, OutT>(         \
        dev_ctx, ins, &outs, -1, binary_func);                           \
  }

DEFINE_LOGICAL_BINARY_KERNEL(And)
DEFINE_LOGICAL_BINARY_KERNEL(Or)
DEFINE_LOGICAL_BINARY_KERNEL(Xor)
#undef DEFINE_LOGICAL_BINARY_KERNEL

template <typename T, typename Context>
void LogicalNotKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      DenseTensor* out) {
  using InT = typename funcs::LogicalNotFunctor<T>::ELEMENT_TYPE;
  using OutT = bool;

  dev_ctx.template Alloc<bool>(out);
  funcs::LogicalNotFunctor<T> unary_func;
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  funcs::BroadcastKernel<ElementwiseType::kUnary, InT, OutT>(
      dev_ctx, ins, &outs, -1, unary_func);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    logical_and, KPS, ALL_LAYOUT, phi::LogicalAndKernel, int /*, float*/) {}
PD_REGISTER_KERNEL(
    logical_Or, KPS, ALL_LAYOUT, phi::LogicalOrKernel, int /*, float*/) {}
PD_REGISTER_KERNEL(
    logical_Not, KPS, ALL_LAYOUT, phi::LogicalNotKernel, int /*, float*/) {}
PD_REGISTER_KERNEL(
    logical_Xor, KPS, ALL_LAYOUT, phi::LogicalXorKernel, int /*, float*/) {}
#endif
